#include "hip/hip_runtime.h"
#include <iostream>
#include <boost/program_options.hpp>
#include <omp.h>
#include <new>
#include <nvtx3/nvToolsExt.h>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <iomanip>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>
#include <cub/block/block_store.cuh>
#define OFFSET(x, y, m) (((x) * (m)) + (y))


namespace po = boost::program_options;

// cuda unique_ptr
template<typename T>
using cuda_unique_ptr = std::unique_ptr<T,std::function<void(T*)>>;


void cudaCheck(hipError_t error, char err_src[]) { //error printing function to reduce line count
    if (error != hipSuccess) {
        printf("Error: %i while performing %s \n", error, err_src);
        exit(EXIT_FAILURE);
    }
}
// new
template<typename T>
T* cuda_new(size_t size)
{
    T *d_ptr;
    hipError_t cudaErr = hipSuccess;
    cudaErr = hipMalloc((void **)&d_ptr, sizeof(T) * size);
    cudaCheck(cudaErr, "error during hipMalloc");
    return d_ptr;
}

// delete
template<typename T>
void cuda_delete(T *dev_ptr)
{
    hipFree(dev_ptr);
}


__global__ void subtractArrays(const double *A, const double *Anew, double *Sub_res , int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if ((i >= 0) && (i < m) && (j >= 0) && (j < m)) {
        Sub_res[OFFSET(i,j,m)] = A[OFFSET(i,j,m)] - Anew[OFFSET(i,j,m)];
    }
}


__global__ void calcNext(double *A, double *Anew, int m, bool calcLeft) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (calcLeft){
        if ((i > 0) && (i < m - 1) && (j > 0) && (j < m - 1)) {
            A[OFFSET(j, i, m)] = 0.25 * (Anew[OFFSET(j, i + 1, m)] + Anew[OFFSET(j, i - 1, m)]
            + Anew[OFFSET(j - 1, i, m)] + Anew[OFFSET(j + 1, i, m)]);
        }
    }
    else{
        if ((i > 0) && (i < m - 1) && (j > 0) && (j < m - 1)) {
            Anew[OFFSET(j, i, m)] = 0.25 * (A[OFFSET(j, i + 1, m)] + A[OFFSET(j, i - 1, m)]
            + A[OFFSET(j - 1, i, m)] + A[OFFSET(j + 1, i, m)]);
        }
    }
}



int main(int argc, char **argv)
{
    int m = 256;
    int iter_max = 1000000;
    double tol = 1.0e-6;
    double error = 1.0;
    po::options_description desc("Options");
    desc.add_options()
        ("help", "print help")
        ("error,e", po::value<double>(&tol)->default_value(tol), "min error")
        ("size,n", po::value<int>(&m)->default_value(m), "size of grid")
        ("iterations,i", po::value<int>(&iter_max)->default_value(iter_max), "number of iterations");

    // Парсинг аргументов командной строки
    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);
    int n = m;

    std::unique_ptr<double[]> A_ptr(new double[m*m]);
    std::unique_ptr<double[]> Anew_ptr(new double[m*m]);
    std::unique_ptr<double[]> Subtract_temp_ptr(new double[m*m]);

    double* A = A_ptr.get();
    double* Anew = Anew_ptr.get();
    double* Subtract_temp = Subtract_temp_ptr.get();

    nvtxRangePushA("init");
    memset(A, 0, n * n * sizeof(double));
    memset(Anew, 0, n * n * sizeof(double));

    double corners[4] = {10, 20, 30, 20};
    A[0] = corners[0];
    A[n - 1] = corners[1];
    A[n * n - 1] = corners[2];
    A[n * (n - 1)] = corners[3];
    Anew[0] = corners[0];
    Anew[n - 1] = corners[1];
    Anew[n * n - 1] = corners[2];
    Anew[n * (n - 1)] = corners[3];
    double step = (corners[1] - corners[0]) / (n - 1);


    for (int i = 1; i < n - 1; i ++) {
        A[i] = corners[0] + i * step;
        A[n * i] = corners[0] + i * step;
        A[(n-1) + n * i] = corners[1] + i * step;
        A[n * (n-1) + i] = corners[3] + i * step;
        Anew[i] = corners[0] + i * step;
        Anew[n * i] = corners[0] + i * step;
        Anew[(n-1) + n * i] = corners[1] + i * step;
        Anew[n * (n-1) + i] = corners[3] + i * step;
    }
    nvtxRangePop();

    // размерности grid и block
    dim3 grid(32 , 32);
	dim3 block(n/32, n/32);

    hipError_t cudaErr = hipSuccess;
    hipStream_t stream;
    hipStreamCreate(&stream);
    cudaCheck(cudaErr, "error during hipStreamCreate");

    cuda_unique_ptr<double> d_unique_ptr_error(cuda_new<double>(0), cuda_delete<double>);
    cuda_unique_ptr<void> d_unique_ptr_temp_storage(cuda_new<void>(0), cuda_delete<void>);

    cuda_unique_ptr<double> d_unique_ptr_A(cuda_new<double>(m*m), cuda_delete<double>);
    cuda_unique_ptr<double> d_unique_ptr_Anew(cuda_new<double>(m*m), cuda_delete<double>);
    cuda_unique_ptr<double> d_unique_ptr_Subtract_temp(cuda_new<double>(m*m), cuda_delete<double>);
    
    // выделение памяти и перенос на GPU
	double *d_error_ptr = d_unique_ptr_error.get();
	cudaErr = hipMalloc((void**)&d_error_ptr, sizeof(double));


    double *d_A = d_unique_ptr_A.get();

	double *d_Anew = d_unique_ptr_Anew.get();

    double *d_Subtract_temp = d_unique_ptr_Subtract_temp.get();

    cudaErr = hipMemcpyAsync(d_A, A, m*m*sizeof(double), hipMemcpyHostToDevice, stream);
    cudaCheck(cudaErr, "error during hipMemcpyAsync");
    cudaErr = hipMemcpyAsync(d_Anew, Anew, m*m*sizeof(double), hipMemcpyHostToDevice, stream);
    cudaCheck(cudaErr, "error during hipMemcpyAsync");

	// проверка занимаемой памяти для редукции
    void *d_temp_storage = d_unique_ptr_temp_storage.get();
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_Anew, d_error_ptr, m*m, stream);
    hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
    cudaCheck(cudaErr, "error during hipMalloc");

    printf("temp_storage_bytes: %d\n", temp_storage_bytes);
    printf("Jacobi relaxation Calculation: %d x %d mesh\n", m, m);
    printf("Max iterations: %d\n", iter_max);
    printf("MIN Error: %lf\n\n", tol);

    // graph
    bool graph_created = false;
	hipGraph_t graph;
	hipGraphExec_t instance;

    int iter = 0;
    auto start = std::chrono::high_resolution_clock::now();
    
    nvtxRangePushA("while");
    while (error > tol && iter < iter_max)
    {
        if(!graph_created) {
            // создание графа
            nvtxRangePushA("createGraph");
            cudaErr = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            cudaCheck(cudaErr, "error during hipStreamBeginCapture");
            for (int i = 0; i < 100; i++) {
                calcNext<<<grid, block, 0, stream>>>(d_A, d_Anew, m, (bool)(i % 2));
            }
            cudaErr = hipStreamEndCapture(stream, &graph);
            cudaCheck(cudaErr, "error during hipStreamEndCapture");
            nvtxRangePop();
            cudaErr = hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            cudaCheck(cudaErr, "error during hipGraphInstantiate");
            graph_created = true;
        }
        nvtxRangePushA("startGraph");
        //запуск графа
        cudaErr = hipGraphLaunch(instance, stream);
        cudaCheck(cudaErr, "error during hipGraphLaunch");
        nvtxRangePop(); 
        iter += 100;
        if (iter % 100 == 0){
            nvtxRangePushA("calcError");
            subtractArrays<<<grid, block, 0, stream>>>(d_A, d_Anew, d_Subtract_temp, m);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_Subtract_temp, d_error_ptr, m*m, stream);
            cudaErr = hipMemcpyAsync(&error, d_error_ptr, sizeof(double), hipMemcpyDeviceToHost, stream);
            nvtxRangePop();
        }
        if (iter % 1000 == 0)
            printf("%5d, %0.6f\n", iter, error);
    }
    nvtxRangePop();
    printf("%5d, %0.6f\n", iter, error);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;

    printf("total: %f s\n", elapsed_seconds.count());
    cudaErr = hipMemcpy(A, d_A, m*m*sizeof(double), hipMemcpyDeviceToHost);
    hipGraphDestroy(graph);
    std::ofstream out("out.txt");
    for (int j = 0; j < n; j++){
        for (int i = 0; i < m; i++){
            out << std::left << std::setw(10) << A[OFFSET(j, i, m)] << " ";
        }
        out << std::endl;
    }
    return 0;
}